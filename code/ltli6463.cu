#include "hip/hip_runtime.h"
#include <set>
#include <vector>
#include <chrono>
#include <fstream>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <warpcore/hash_set.cuh>

using UINT_64 = std::uint64_t;

const std::size_t maxNumOfTraces = 64;

__constant__ char d_traceLen[maxNumOfTraces];

inline
hipError_t checkCuda(hipError_t res) {
#ifndef MEASUREMENT_MODE
    if (res != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(res));
        assert(res == hipSuccess);
    }
#endif
    return res;
}

// Finding the left and right indices that makes the final LTL to bring to the host later
__global__ void generateResIndices(
    const int index,
    const int alphabetSize,
    const int *d_leftIdx,
    const int *d_rightIdx,
    int *d_FinalLTLIdx)
{

    int resIdx = 0;
    while (d_FinalLTLIdx[resIdx] != -1) resIdx++;
    int queue[600];
    queue[0] = index;
    int head = 0;
    int tail = 1;
    while (head < tail) {
        int ltl = queue[head];
        int l = d_leftIdx[ltl];
        int r = d_rightIdx[ltl];
        d_FinalLTLIdx[resIdx++] = ltl;
        d_FinalLTLIdx[resIdx++] = l;
        d_FinalLTLIdx[resIdx++] = r;
        if (l >= alphabetSize) queue[tail++] = l;
        if (r >= alphabetSize) queue[tail++] = r;
        head++;
    }

}

__device__ void makeRlxUnqChkCSs(
    UINT_64 *CS,
    UINT_64 &hCS,
    UINT_64 &lCS,
    const int numOfTraces,
    const int RlxUnqChkTyp,
    const int lenSum)
{

    if (lenSum > 126) {

        // we need an relaxed uniqueness check

        switch (RlxUnqChkTyp) {

            case 1: {

                const int stride = lenSum / 126;

                int j = 0;
                for (int i = 0; i < numOfTraces; ++i) {
                    for (int k = 0; k < d_traceLen[i]; k += stride, ++j) {
                        if (j < 63) {
                            if (CS[i] & ((UINT_64) 1 << k)) lCS |= (UINT_64) 1 << j;
                        } else if (j < 126) {
                            if (CS[i] & ((UINT_64) 1 << k)) hCS |= (UINT_64) 1 << (j - 63);
                        } else break;
                    }
                }

                break;
            }

            case 2: {

                int j = 0;
                for (int i = 0; i < numOfTraces; ++i) {
                    UINT_64 bitPtr = 1;
                    int maxbitsForThisTrace = (126 * d_traceLen[i] + lenSum) / lenSum;
                    for (int k = 0; k < maxbitsForThisTrace; ++k, ++j, bitPtr <<= 1) {
                        if (j < 63) {
                            if (CS[i] & bitPtr) lCS |= (UINT_64) 1 << j;
                        } else if (j < 126) {
                            if (CS[i] & bitPtr) hCS |= (UINT_64) 1 << (j - 63);
                        } else break;
                    }
                }

                break;
            }

            case 3: {

                for (int i = 0; i < numOfTraces; ++i) {
                    UINT_64 x = CS[i];
                    x = (x ^ (x >> 30)) * UINT64_C(0xbf58476d1ce4e5b9);
                    x = (x ^ (x >> 27)) * UINT64_C(0x94d049bb133111eb);
                    x = x ^ (x >> 31);
                    if (i < numOfTraces / 2) hCS ^= x; else lCS ^= x;
                }

                break;
            }

        }

    } else {

        // the result will be minimal

        int j = 0;
        for (int i = 0; i < numOfTraces; ++i) {
            UINT_64 bitPtr = 1;
            for (int k = 0; k < d_traceLen[i]; ++k, ++j, bitPtr <<= 1) {
                if (j < 63) {
                    if (CS[i] & bitPtr) lCS |= (UINT_64) 1 << j;
                } else if (j < 126) {
                    if (CS[i] & bitPtr) hCS |= (UINT_64) 1 << (j - 63);
                } else break;
            }
        }

    }

}

// Initialising the hashSets with the alphabet before starting the enumeration
template<class hash_set_t>
__global__ void hashSetsInitialisation(
    const int numOfTraces,
    const int RlxUnqChkTyp,
    const int lenSum,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    UINT_64 *d_LTLcache)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    UINT_64 CS[maxNumOfTraces];

    for (int i = 0; i < numOfTraces; ++i)
        CS[i] = d_LTLcache[tid * numOfTraces + i];

    UINT_64 hCS{}, lCS{};
    makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

    const auto group = warpcore::cg::tiled_partition <1>
    (warpcore::cg::this_thread_block());
    int H = cHashSet.insert(hCS, group);
    int L = cHashSet.insert(lCS, group);
    H = (H > 0) ? H : -H;
    L = (L > 0) ? L : -L;
    UINT_64 HL = H; HL <<= 32; HL |= L;
    iHashSet.insert(HL, group);

}

// Generating ~p for p in indices between idx1 and idx2 in the LTLcache
template<class hash_set_t>
__global__ void Negation(
    const int idx1,
    const int idx2,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < idx2 - idx1 + 1) {

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i) {
            UINT_64 negationFixer = ((UINT_64) 1 << d_traceLen[i]) - 1;
            CS[i] = ~d_LTLcache[(idx1 + tid) * numOfTraces + i] & negationFixer;
        }

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0; // just to avoid getting removed

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating phi1 & phi2 (intersection)
// For phi1 in indices between idx1 and idx2 in the LTLcache
// For phi2 in indices between idx3 and idx4 in the LTLcache
template<class hash_set_t>
__global__ void Intersection(
    const int idx1,
    const int idx2,
    const int idx3,
    const int idx4,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < (idx4 - idx3 + 1) * (idx2 - idx1 + 1)) {

        int ldx = idx1 + tid / (idx4 - idx3 + 1);
        int rdx = idx3 + tid % (idx4 - idx3 + 1);

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i)
            CS[i] = d_LTLcache[ldx * numOfTraces + i] & d_LTLcache[rdx * numOfTraces + i];

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = ldx;
                d_temp_rightIdx[tid] = rdx;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = ldx;
                d_temp_rightIdx[tid] = rdx;

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating phi1 | phi2 (union)
// For phi1 in indices between idx1 and idx2 in the LTLcache
// For phi2 in indices between idx3 and idx4 in the LTLcache
template<class hash_set_t>
__global__ void Union(
    const int idx1,
    const int idx2,
    const int idx3,
    const int idx4,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < (idx4 - idx3 + 1) * (idx2 - idx1 + 1)) {

        int ldx = idx1 + tid / (idx4 - idx3 + 1);
        int rdx = idx3 + tid % (idx4 - idx3 + 1);

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i)
            CS[i] = d_LTLcache[ldx * numOfTraces + i] | d_LTLcache[rdx * numOfTraces + i];

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = ldx;
                d_temp_rightIdx[tid] = rdx;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = ldx;
                d_temp_rightIdx[tid] = rdx;

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating X(phi) for phi in indices between idx1 and idx2 in the LTLcache
template<class hash_set_t>
__global__ void Next(
    const int idx1,
    const int idx2,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < idx2 - idx1 + 1) {

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[(idx1 + tid) * numOfTraces + i] >> 1;
        }

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0; // just to avoid getting removed

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating F(phi) for phi in indices between idx1 and idx2 in the LTLcache
template<class hash_set_t>
__global__ void Finally(
    const int idx1,
    const int idx2,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < idx2 - idx1 + 1) {

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[(idx1 + tid) * numOfTraces + i];
            CS[i] |= CS[i] >> 1;  CS[i] |= CS[i] >> 2;
            CS[i] |= CS[i] >> 4;  CS[i] |= CS[i] >> 8;
            CS[i] |= CS[i] >> 16; CS[i] |= CS[i] >> 32;
        }

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0; // just to avoid getting removed

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating G(phi) for phi in indices between idx1 and idx2 in the LTLcache
template<class hash_set_t>
__global__ void Globally(
    const int idx1,
    const int idx2,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < idx2 - idx1 + 1) {

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[(idx1 + tid) * numOfTraces + i];
            UINT_64 cs = ~CS[i] & (((UINT_64) 1 << d_traceLen[i]) - 1);
            cs |= cs >> 1;  cs |= cs >> 2;
            cs |= cs >> 4;  cs |= cs >> 8;
            cs |= cs >> 16; cs |= cs >> 32;
            CS[i] &= ~cs;
        }

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

            if (found) {
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0;
                atomicCAS(d_FinalLTLIdx, -1, tid);
            }

        } else {

            UINT_64 hCS{}, lCS{};
            makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H = cHashSet.insert(hCS, group);
            int L = cHashSet.insert(lCS, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            UINT_64 HL = H; HL <<= 32; HL |= L;
            bool CS_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
                d_temp_leftIdx[tid] = idx1 + tid;
                d_temp_rightIdx[tid] = 0; // just to avoid getting removed

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid] = -1;
                d_temp_rightIdx[tid] = -1;

            }

        }

    }

}

// Generating phi1 U phi2, phi2 U phi1 (Until)
// For phi1 in indices between idx1 and idx2 in the LTLcache
// For phi2 in indices between idx3 and idx4 in the LTLcache
template<class hash_set_t>
__global__ void Until(
    const int idx1,
    const int idx2,
    const int idx3,
    const int idx4,
    const int numOfP,
    const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx,
    hash_set_t cHashSet,
    hash_set_t iHashSet,
    int *d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < (idx4 - idx3 + 1) * (idx2 - idx1 + 1)) {

        int ldx = idx1 + tid / (idx4 - idx3 + 1);
        int rdx = idx3 + tid % (idx4 - idx3 + 1);

        const int numOfTraces = numOfP + numOfN;

        UINT_64 CS1[maxNumOfTraces], CS2[maxNumOfTraces];

        for (int i = 0; i < numOfTraces; ++i) {

            UINT_64 l = d_LTLcache[ldx * numOfTraces + i];
            UINT_64 r = d_LTLcache[rdx * numOfTraces + i];

            UINT_64 x = l;
            UINT_64 y = r;

            y |= x & (y >> 1);
            x &= x >> 1;
            y |= x & (y >> 2);
            x &= x >> 2;
            y |= x & (y >> 4);
            x &= x >> 4;
            y |= x & (y >> 8);
            x &= x >> 8;
            y |= x & (y >> 16);
            x &= x >> 16;
            y |= x & (y >> 32);

            CS1[i] = y;

            x = r;
            y = l;

            y |= x & (y >> 1);
            x &= x >> 1;
            y |= x & (y >> 2);
            x &= x >> 2;
            y |= x & (y >> 4);
            x &= x >> 4;
            y |= x & (y >> 8);
            x &= x >> 8;
            y |= x & (y >> 16);
            x &= x >> 16;
            y |= x & (y >> 32);

            CS2[i] = y;

        }

        if (onTheFly) {

            bool found = true;
            for (int i = 0; found && i < numOfP; ++i) if (!(CS1[i] & 1)) found = false;
            for (int i = numOfP; found && i < numOfTraces; ++i) if (CS1[i] & 1) found = false;

            if (found) {

                atomicCAS(d_FinalLTLIdx, -1, tid * 2);
                d_temp_leftIdx[tid * 2] = ldx;
                d_temp_rightIdx[tid * 2] = rdx;

            } else {

                found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS2[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS2[i] & 1) found = false;

                if (found) {
                    atomicCAS(d_FinalLTLIdx, -1, tid * 2 + 1);
                    d_temp_leftIdx[tid * 2 + 1] = rdx;
                    d_temp_rightIdx[tid * 2 + 1] = ldx;
                }

            }

        } else {

            UINT_64 hCS1{}, lCS1{};
            makeRlxUnqChkCSs(CS1, hCS1, lCS1, numOfTraces, RlxUnqChkTyp, lenSum);

            UINT_64 hCS2{}, lCS2{};
            makeRlxUnqChkCSs(CS2, hCS2, lCS2, numOfTraces, RlxUnqChkTyp, lenSum);

            const auto group = warpcore::cg::tiled_partition <1>
            (warpcore::cg::this_thread_block());
            int H, L; UINT_64 HL;

            H = cHashSet.insert(hCS1, group);
            L = cHashSet.insert(lCS1, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            HL = H; HL <<= 32; HL |= L;
            bool CS1_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            H = cHashSet.insert(hCS2, group);
            L = cHashSet.insert(lCS2, group);
            H = (H > 0) ? H : -H;
            L = (L > 0) ? L : -L;
            HL = H; HL <<= 32; HL |= L;
            bool CS2_is_unique = (iHashSet.insert(HL, group) > 0) ? false : true;

            if (CS1_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[(tid * 2) * numOfTraces + i] = CS1[i];
                d_temp_leftIdx[tid * 2] = ldx;
                d_temp_rightIdx[tid * 2] = rdx;

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS1[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS1[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid * 2);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[(tid * 2) * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid * 2] = -1;
                d_temp_rightIdx[tid * 2] = -1;

            }

            if (CS2_is_unique) {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[(tid * 2 + 1) * numOfTraces + i] = CS2[i];
                d_temp_leftIdx[tid * 2 + 1] = rdx;
                d_temp_rightIdx[tid * 2 + 1] = ldx;

                bool found = true;
                for (int i = 0; found && i < numOfP; ++i) if (!(CS2[i] & 1)) found = false;
                for (int i = numOfP; found && i < numOfTraces; ++i) if (CS2[i] & 1) found = false;
                if (found) atomicCAS(d_FinalLTLIdx, -1, tid * 2 + 1);

            } else {

                for (int i = 0; i < numOfTraces; ++i)
                    d_temp_LTLcache[(tid * 2 + 1) * numOfTraces + i] = (UINT_64) -1;
                d_temp_leftIdx[tid * 2 + 1] = -1;
                d_temp_rightIdx[tid * 2 + 1] = -1;

            }

        }

    }

}

// Generating the final RE string recursively
// When all the left and right indices are ready in the host
std::string toString(
    int index,
    std::map<int, std::pair<int, int>> &indicesMap,
    const std::set<char> &alphabet,
    const int *startPoints)
{

    if (index < alphabet.size()) {
        std::string s(1, *next(alphabet.begin(), index));
        return s;
    }
    int i = 0;
    while (index >= startPoints[i]){i++;}
    i--;

    if (i % 7 == 0) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "~(" + res + ")";
    }

    if (i % 7 == 1) {
        std::string left  = toString(indicesMap[index].first,  indicesMap, alphabet, startPoints);
        std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
        return "(" + left + ")" + "&" + "(" + right + ")";
    }

    if (i % 7 == 2) {
        std::string left  = toString(indicesMap[index].first,  indicesMap, alphabet, startPoints);
        std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
        return "(" + left + ")" + "|" + "(" + right + ")";
    }

    if (i % 7 == 3) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "X(" + res + ")";
    }

    if (i % 7 == 4) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "F(" + res + ")";
    }

    if (i % 7 == 5) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "G(" + res + ")";
    }

    std::string left  = toString(indicesMap[index].first,  indicesMap, alphabet, startPoints);
    std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
    return "(" + left + ")" + "U" + "(" + right + ")";

}

// Bringing the left and right indices of the LTL from device to host
// If LTL is found, this index is from the temp memory               (temp = true)
// For printing other LTLs if needed, indices are in the main memory (temp = false)
std::string LTLtoString (
    bool temp,
    const int FinalLTLIdx,
    const int lastIdx,
    const std::set<char> &alphabet,
    const int *startPoints,
    const int *d_leftIdx,
    const int *d_rightIdx,
    const int *d_temp_leftIdx,
    const int *d_temp_rightIdx)
{

    auto *LIdx = new int [1];
    auto *RIdx = new int [1];

    if (temp) {
        checkCuda( hipMemcpy(LIdx, d_temp_leftIdx  + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
        checkCuda( hipMemcpy(RIdx, d_temp_rightIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
    } else {
        checkCuda( hipMemcpy(LIdx, d_leftIdx +  FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
        checkCuda( hipMemcpy(RIdx, d_rightIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
    }

    auto alphabetSize = static_cast<int> (alphabet.size());

    int *d_resIndices;
    checkCuda( hipMalloc(&d_resIndices, 600 * sizeof(int)) );

    thrust::device_ptr<int> d_resIndices_ptr(d_resIndices);
    thrust::fill(d_resIndices_ptr, d_resIndices_ptr + 600, -1);

    if (*LIdx >= alphabetSize) generateResIndices<<<1, 1>>>(*LIdx, alphabetSize, d_leftIdx, d_rightIdx, d_resIndices);
    if (*RIdx >= alphabetSize) generateResIndices<<<1, 1>>>(*RIdx, alphabetSize, d_leftIdx, d_rightIdx, d_resIndices);

    int resIndices[600];
    checkCuda( hipMemcpy(resIndices, d_resIndices, 600 * sizeof(int), hipMemcpyDeviceToHost) );

    std::map<int, std::pair<int, int>> indicesMap;

    if (temp) indicesMap.insert(std::make_pair(INT_MAX - 1, std::make_pair(*LIdx, *RIdx)));
    else      indicesMap.insert(std::make_pair(FinalLTLIdx, std::make_pair(*LIdx, *RIdx)));

    int i = 0;
    while (resIndices[i] != -1 && i + 2 < 600) {
        int ltl = resIndices[i];
        int l = resIndices[i + 1];
        int r = resIndices[i + 2];
        indicesMap.insert( std::make_pair(ltl,  std::make_pair(l, r)));
        i += 3;
    }

    if (i + 2 >= 600) return "Size of the output is too big";

    hipFree(d_resIndices);

    if (temp) return toString(INT_MAX - 1, indicesMap, alphabet, startPoints);
    else      return toString(FinalLTLIdx, indicesMap, alphabet, startPoints);

}

// Transfering the unique CSs from temp to main LTLcache
void storeUniqueLTLs(
    int N,
    int &lastIdx,
    const int numOfTraces,
    const int LTLcacheCapacity,
    bool &onTheFly,
    UINT_64 *d_LTLcache,
    UINT_64 *d_temp_LTLcache,
    int *d_leftIdx,
    int *d_rightIdx,
    int *d_temp_leftIdx,
    int *d_temp_rightIdx)
{

    thrust::device_ptr<UINT_64> new_end_ptr;
    thrust::device_ptr<UINT_64> d_LTLcache_ptr(d_LTLcache + numOfTraces * lastIdx);
    thrust::device_ptr<UINT_64> d_temp_LTLcache_ptr(d_temp_LTLcache);
    thrust::device_ptr<int> d_leftIdx_ptr(d_leftIdx + lastIdx);
    thrust::device_ptr<int> d_rightIdx_ptr(d_rightIdx + lastIdx);
    thrust::device_ptr<int> d_temp_leftIdx_ptr(d_temp_leftIdx);
    thrust::device_ptr<int> d_temp_rightIdx_ptr(d_temp_rightIdx);

    new_end_ptr = // end of d_temp_LTLcache
    thrust::remove(d_temp_LTLcache_ptr, d_temp_LTLcache_ptr + numOfTraces * N, (UINT_64) -1);
    thrust::remove(d_temp_leftIdx_ptr,  d_temp_leftIdx_ptr  + N, -1);
    thrust::remove(d_temp_rightIdx_ptr, d_temp_rightIdx_ptr + N, -1);

    // It stores all (or a part of) unique CSs until language cahce gets full
    // If language cache gets full, it makes onTheFly mode on
    int numberOfNewUniqueLTLs = static_cast<int>(new_end_ptr - d_temp_LTLcache_ptr) / numOfTraces;
    if (lastIdx + numberOfNewUniqueLTLs > LTLcacheCapacity) {
        N = LTLcacheCapacity - lastIdx;
        onTheFly = true;
    } else N = numberOfNewUniqueLTLs;

    thrust::copy_n(d_temp_LTLcache_ptr, numOfTraces * N, d_LTLcache_ptr);
    thrust::copy_n(d_temp_leftIdx_ptr,  N, d_leftIdx_ptr);
    thrust::copy_n(d_temp_rightIdx_ptr, N, d_rightIdx_ptr);

    lastIdx += N;

}

int costOf(const int index, const int *startPoints) {
    int i = 0;
    while (index >= startPoints[i]) {i++;}
    return((i - 1) / 4);
}

std::string LTLI(
    const unsigned short *costFun, 
    const unsigned short maxCost, 
    const unsigned int RlxUnqChkTyp,
    const unsigned int NegType, 
    const std::set<char> alphabet, 
    int &LTLcost, 
    std::uint64_t &allLTLs,
    const std::vector<std::vector<std::string>> pos, 
    const std::vector<std::vector<std::string>> neg) {

    // --------------------------------
    // Generating and checking alphabet
    // --------------------------------

    const int numOfP = pos.size();
    const int numOfN = neg.size();
    const int numOfTraces = numOfP + numOfN;

    int maxLenOfTraces{};
    auto *traceLen = new char [numOfTraces];

    int TLIdx{};
    int lenSum{};
    for (const auto &trace : pos) {
        lenSum += trace.size();
        traceLen[TLIdx++] = trace.size();
        if (trace.size() > maxLenOfTraces) maxLenOfTraces = trace.size();
    }
    for (const auto &trace : neg) {
        lenSum += trace.size();
        traceLen[TLIdx++] = trace.size();
        if (trace.size() > maxLenOfTraces) maxLenOfTraces = trace.size();
    }

    if (numOfTraces > maxNumOfTraces || maxLenOfTraces > sizeof(UINT_64) * 8 - 1) {
        printf("In this version, The input can have:\n");
        printf("1) At most %zu traces. It is currently %d.\n", maxNumOfTraces, numOfTraces);
        printf("2) Max(len(trace)) = %d. It is currently %d.\n", static_cast<int>(sizeof(UINT_64) * 8 - 1), maxLenOfTraces);
        return "see_the_error";
    }

    // Copying the length of traces into the constant memory
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_traceLen), traceLen, numOfTraces * sizeof(char)) );

    const int alphabetSize = static_cast<int>(alphabet.size());

    auto *LTLcache = new UINT_64 [alphabetSize * numOfTraces];

    // Index of the last free position in the LTLcache
    int lastIdx{};

    #ifndef MEASUREMENT_MODE
        printf("Cost %-2d | (A) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                costFun[0], allLTLs, 0, alphabetSize);
    #endif

    int index{};
    for (int i = 0; i < alphabetSize; ++i) {
        bool found = true;
        std::string ch(1, *next(alphabet.begin(), i));
        for (const auto &trace : pos) {
            UINT_64 binTrace{};
            UINT_64 idx = 1;
            for (const auto &token : trace) {
                for (const auto &c : token) {
                    if (c == ch[0]) {
                        binTrace |= idx;
                        break;
                    }
                }
                idx <<= 1;
            }
            LTLcache[index++] = binTrace;
            if (!(binTrace & 1)) found = false;
        }
        for (const auto &trace : neg) {
            UINT_64 binTrace{};
            UINT_64 idx = 1;
            for (const auto &token : trace) {
                for (const auto &c : token) {
                    if (c == ch[0]) {
                        binTrace |= idx;
                        break;
                    }
                }
                idx <<= 1;
            }
            LTLcache[index++] = binTrace;
            if (binTrace & 1) found = false;
        }
        allLTLs++; lastIdx++;
        if (found) return ch;
    }

    // ------------------------------------
    // Memory allocation and initialisation
    // ------------------------------------

    // cost function
    int c1 = costFun[0]; // cost of p
    int c2 = costFun[1]; // cost of ~
    int c3 = costFun[2]; // cost of &
    int c4 = costFun[3]; // cost of |
    int c5 = costFun[4]; // cost of X
    int c6 = costFun[5]; // cost of F
    int c7 = costFun[6]; // cost of G
    int c8 = costFun[7]; // cost of U

    int maxAllocationSize;
    hipDeviceGetAttribute(&maxAllocationSize, hipDeviceAttributeMaxPitch, 0);

    const int LTLcacheCapacity = maxAllocationSize / (numOfTraces * sizeof(UINT_64)) * 1.5;
    const int temp_LTLcacheCapacity = LTLcacheCapacity / 2;

    // const int LTLcacheCapacity = 2000000;
    // const int temp_LTLcacheCapacity = 100000000;

    // 7 for ~, &, |, X, F, G, U
    int *startPoints = new int [(maxCost + 2) * 7]();
    startPoints[c1 * 7 + 6] = lastIdx;
    startPoints[(c1 + 1) * 7] = lastIdx;

    int *d_FinalLTLIdx;
    auto *FinalLTLIdx = new int [1]; *FinalLTLIdx = -1;
    checkCuda( hipMalloc(&d_FinalLTLIdx, sizeof(int)) );
    checkCuda( hipMemcpy(d_FinalLTLIdx, FinalLTLIdx, sizeof(int), hipMemcpyHostToDevice) );

    UINT_64 *d_LTLcache, *d_temp_LTLcache;
    int *d_leftIdx, *d_rightIdx, *d_temp_leftIdx, *d_temp_rightIdx;
    checkCuda( hipMalloc(&d_leftIdx, LTLcacheCapacity * sizeof(int)) );
    checkCuda( hipMalloc(&d_rightIdx, LTLcacheCapacity * sizeof(int)) );
    checkCuda( hipMalloc(&d_temp_leftIdx, temp_LTLcacheCapacity * sizeof(int)) );
    checkCuda( hipMalloc(&d_temp_rightIdx, temp_LTLcacheCapacity * sizeof(int)) );
    checkCuda( hipMalloc(&d_LTLcache, LTLcacheCapacity * numOfTraces * sizeof(UINT_64)) );
    checkCuda( hipMalloc(&d_temp_LTLcache, temp_LTLcacheCapacity * numOfTraces * sizeof(UINT_64)) );

    using hash_set_t = warpcore::HashSet<
    UINT_64,         // key type
    UINT_64(0) - 1,  // empty key
    UINT_64(0) - 2,  // tombstone key
    warpcore::probing_schemes::QuadraticProbing<warpcore::hashers::MurmurHash <UINT_64>>>;

    hash_set_t cHashSet(2 * LTLcacheCapacity);
    hash_set_t iHashSet(2 * LTLcacheCapacity);

    checkCuda( hipMemcpy(d_LTLcache, LTLcache, alphabetSize * numOfTraces * sizeof(UINT_64), hipMemcpyHostToDevice) );
    hashSetsInitialisation<hash_set_t><<<1, alphabetSize>>>(numOfTraces, RlxUnqChkTyp, lenSum, cHashSet, iHashSet, d_LTLcache);

    // ----------------------------
    // Enumeration of the next LTLs
    // ----------------------------

    bool onTheFly = false, lastRound = false;
    int shortageCost = -1;

    for (LTLcost = c1 + 1; LTLcost <= maxCost; ++LTLcost) {


        // Once it uses a previous cost that is not fully stored, it should continue as the last round
	    if (onTheFly) {
            int dif = LTLcost - shortageCost;
            if (dif == c2 || dif == c1 + c3 || dif == c1 + c4 || dif == c5 || dif == c6 || dif == c7 || dif == c1 + c8) lastRound = true;
        }


        // negation (~)
        // NegType = 1 is for negation of phi
        // NegType = 2 is for negation of char only
        if ((NegType == 1 && LTLcost - c2 >= c1) || (NegType == 2 && LTLcost - c2 == c1))
        if (LTLcost - c2 >= c1) {

            int Idx1 = startPoints[(LTLcost - c2) * 7];
            int Idx2 = startPoints[(LTLcost - c2 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (~) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Negation<hash_set_t><<<Blc, 1024>>>(x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                        d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 1] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 1] = lastIdx;

        // intersection (&)
        for (int i = c1; 2 * i <= LTLcost - c3; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c3) * 7];
            int Idx4 = startPoints[(LTLcost - i - c3 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (Idx2 - Idx1 + 1) - 1, Idx4 - x);
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (&) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Intersection<hash_set_t><<<Blc, 1024>>>(Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                            d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 2] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[LTLcost * 7 + 2] = lastIdx;

        // union (|)
        for (int i = c1; 2 * i <= LTLcost - c4; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c4) * 7];
            int Idx4 = startPoints[(LTLcost - i - c4 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (Idx2 - Idx1 + 1) - 1, Idx4 - x);
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (|) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Union<hash_set_t><<<Blc, 1024>>>(Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                     d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 3] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[LTLcost * 7 + 3] = lastIdx;

        // next (X)
        if (LTLcost - c5 >= c1) {

            int Idx1 = startPoints[(LTLcost - c5) * 7];
            int Idx2 = startPoints[(LTLcost - c5 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (X) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Next<hash_set_t><<<Blc, 1024>>>(x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                    d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 4] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 4] = lastIdx;

        // finally (F)
        if (LTLcost - c6 >= c1) {

            int Idx1 = startPoints[(LTLcost - c6) * 7];
            int Idx2 = startPoints[(LTLcost - c6 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (F) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Finally<hash_set_t><<<Blc, 1024>>>(x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                       d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 5] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 5] = lastIdx;

        // globally (G)
        if (LTLcost - c7 >= c1) {

            int Idx1 = startPoints[(LTLcost - c7) * 7];
            int Idx2 = startPoints[(LTLcost - c7 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (G) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Globally<hash_set_t><<<Blc, 1024>>>(x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                        d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) {startPoints[LTLcost * 7 + 6] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 6] = lastIdx;

        // until (U)
        for (int i = c1; 2 * i <= LTLcost - c8; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c8) * 7];
            int Idx4 = startPoints[(LTLcost - i - c8 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (2 * (Idx2 - Idx1 + 1)) - 1, Idx4 - x); // 2 is for until only (lUr and rUl)
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
                    #ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (U) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                                LTLcost, allLTLs, lastIdx, 2 * N);
                    #endif
                    int Blc = (N + 1023) / 1024;
                    Until<hash_set_t><<<Blc, 1024>>>(Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                     d_temp_leftIdx, d_temp_rightIdx, cHashSet, iHashSet, d_FinalLTLIdx);
                    checkCuda( hipPeekAtLastError() );
                    checkCuda( hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost) );
                    allLTLs += 2 * N;
                    if (*FinalLTLIdx != -1) {startPoints[(LTLcost + 1) * 7] = INT_MAX; goto exitEnumeration;}
                    if (!onTheFly) storeUniqueLTLs(2 * N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                                                   d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[(LTLcost + 1) * 7] = lastIdx;

        if (lastRound) break;
        if (onTheFly && shortageCost == -1) shortageCost = LTLcost;

    }

    if (LTLcost == maxCost + 1) LTLcost--;

    exitEnumeration:

    std::string output;
    bool isLTLFromTempLTLcache = true;

    if (*FinalLTLIdx != -1) {

        output = LTLtoString(isLTLFromTempLTLcache, *FinalLTLIdx, lastIdx, alphabet, startPoints,
        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);

    } else {

        output = "not_found";

    }

    /*
    const int NNN = 1000;
    auto *cache2 = new UINT_64[numOfTraces * NNN];
    auto *left_indices = new int[NNN];
    auto *right_indices = new int[NNN];
    checkCuda( hipMemcpy(cache2, d_LTLcache, numOfTraces * NNN * sizeof(UINT_64), hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(left_indices, d_leftIdx, NNN * sizeof(int), hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(right_indices, d_rightIdx, NNN * sizeof(int), hipMemcpyDeviceToHost) );

    for (int i = 0; i < NNN; ++i) {
        std::cout << i << " --> \t \t";
        std::string out = LTLtoString(false, i, lastIdx, alphabet, startPoints,
        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
        std::cout << out << "\t \t";
        for (int j = 0; j < numOfTraces; ++j) {
            std::cout << cache2[i * numOfTraces + j] << "\t";
        }
        std::cout << std::endl;
    }
    */

    // cleanup
    hipFree(d_LTLcache);
    hipFree(d_FinalLTLIdx);
    hipFree(d_temp_LTLcache);
    hipFree(d_leftIdx);
    hipFree(d_rightIdx);
    hipFree(d_temp_leftIdx);
    hipFree(d_temp_rightIdx);

    return output;

}

// Reading the input file
bool readFile(
        const std::string& fileName,
        std::set<char> &alphabet,
        std::vector<std::vector<std::string>> &pos,
        std::vector<std::vector<std::string>> &neg)
{

    std::ifstream file(fileName);
    if (file.is_open()) {
        file.seekg(0, std::ios::end);
        std::string line;
        char ch;
        bool foundNewline = false;
        while (!foundNewline && file.tellg() > 0) {
            file.seekg(-2, std::ios::cur);
            file.get(ch);
            if (ch == '\n') foundNewline = true;
        }
        std::getline(file, line);
        std::string alpha;
        for (auto &c : line)
            if (c >= 'a' && c <= 'z') {
                alphabet.insert(c);
                alpha += c;
            }
        file.seekg(0, std::ios::beg);
        while (std::getline(file, line)) {
            std::vector<std::string> trace;
            if (line != "---") {
                std::string token;
                int j{};
                for (auto &c : line) {
                    if (c == ';') {
                        trace.push_back(token);
                        token = "";
                        j = 0;
                    }
                    else if (c == ',') continue;
                    else {
                        if (c == '1') token += alpha[j];
                        j++;
                    }
                }
                trace.push_back(token);
                pos.push_back(trace);
            } else break;
        }
        while (std::getline(file, line)) {
            std::vector<std::string> trace;
            if (line != "---") {
                std::string token;
                int j{};
                for (auto &c : line) {
                    if (c == ';') {
                        trace.push_back(token);
                        token = "";
                        j = 0;
                    } else if (c == ',') continue;
                    else {
                        if (c == '1') token += alpha[j];
                        j++;
                    }
                }
                trace.push_back(token);
                neg.push_back(trace);
            } else break;
        }
        file.close();
        return true;
    } else printf("Failed to open the input file.\n");

    return false;

}

int main (int argc, char *argv[]) {

    // -----------------
    // Reading the input
    // -----------------

    if (argc != 13) {
        printf("Arguments should be in the form of\n");
        printf("-----------------------------------------------------------------\n");
        printf("%s <input_file_address> <c1> <c2> <c3> <c4> <c5> <c6> <c7> <c8> <maxCost> <RlxUnqChkTyp> <NegType>\n", argv[0]);
        printf("-----------------------------------------------------------------\n");
        printf("\nFor example\n");
        printf("-----------------------------------------------------------------\n");
        printf("%s ./input.txt 1 1 1 1 1 1 1 1 500 3 2\n", argv[0]);
        printf("-----------------------------------------------------------------\n");
        return 0;
    }

    bool argError = false;
    for (int i = 2; i < argc - 2; ++i) {
        if (std::atoi(argv[i]) <= 0 || std::atoi(argv[i]) > SHRT_MAX) {
            printf("Argument number %d, \"%s\", should be a positive short integer.\n", i, argv[i]);
            argError = true;
        }
    }
    if (std::atoi(argv[11]) < 1 || std::atoi(argv[11]) > 3) {
        printf("Argument number 11, RlxUnqChkTyp = \"%s\", should be 1, 2, or 3.\n", argv[11]);
        argError = true;
    }
    if (std::atoi(argv[12]) < 1 || std::atoi(argv[12]) > 2) {
        printf("Argument number 12, NegType = \"%s\", should be 1, or 2.\n", argv[12]);
        argError = true;
    }

    if (argError) return 0;

    std::string fileName = argv[1];
    std::set<char> alphabet;
    std::vector<std::vector<std::string>> pos, neg;
    if (!readFile(fileName, alphabet, pos, neg)) return 0;
    unsigned short costFun[8];
    for (int i = 0; i < 8; i++)
        costFun[i] = std::atoi(argv[i + 2]);
    unsigned short maxCost = std::atoi(argv[10]);
    unsigned int RlxUnqChkTyp = std::atoi(argv[11]);
    unsigned int NegType = std::atoi(argv[12]);

    // --------------------------------------
    // Linear Temporal Logic Inference (LTLI)
    // --------------------------------------

    #ifdef MEASUREMENT_MODE
        auto start = std::chrono::high_resolution_clock::now();
    #endif

    std::uint64_t allLTLs{}; int LTLcost = costFun[0];
    std::string output = LTLI(costFun, maxCost, RlxUnqChkTyp, NegType, alphabet, LTLcost, allLTLs, pos, neg);
    if (output == "see_the_error") return 0;

    #ifdef MEASUREMENT_MODE
        auto stop = std::chrono::high_resolution_clock::now();
    #endif

    // -------------------
    // Printing the output
    // -------------------

    printf("\nPositive: \n");
    for (const auto& trace : pos) {
        printf("\t");
        for (const auto& t : trace) {
            std::string s;
            for (const auto& c : t) {
                s += c; s += ", ";
            }
            printf("{%s}\t", s.substr(0, s.length() - 2).c_str());
        }
        printf("\n");
    }

    printf("\nNegative: \n");
    for (const auto& trace : neg) {
        printf("\t");
        for (const auto& t : trace) {
            std::string s;
            for (const auto& c : t) {
                s += c; s += ", ";
            }
            printf("{%s}\t", s.substr(0, s.length() - 2).c_str());
        }
        printf("\n");
    }

    printf("\nCost Function: p:%u, ~:%u, &:%u, |:%u, X:%u, F:%u, G:%u, U:%u",
           costFun[0], costFun[1], costFun[2], costFun[3], costFun[4], costFun[5], costFun[6], costFun[7]);
    printf("\nNumber of Traces: %d", static_cast<int>(pos.size() + neg.size()));
    #ifdef MEASUREMENT_MODE
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
        printf("\nNumber of All LTLs: %lu", allLTLs);
        printf("\nCost of Final LTL: %d", LTLcost);
        printf("\nRunning Time: %f s", (double) duration * 0.000001);
    #endif
    printf("\n\nLTL: \"%s\"\n", output.c_str());

    return 0;

}